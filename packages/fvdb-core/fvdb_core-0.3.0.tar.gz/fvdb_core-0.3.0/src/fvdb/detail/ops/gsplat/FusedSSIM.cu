#include "hip/hip_runtime.h"
// This file contains source code from the fused-ssim library obtained from
// https://github.com/rahul-goel/fused-ssim. The fused-ssim library is licensed under the MIT
// License. Refer to ORSB 5512107 for more. Original license text follows.

// Copyright (c) 2024 Rahul Goel
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: Apache-2.0
//
#include <fvdb/detail/ops/gsplat/FusedSSIM.h>
#include <fvdb/detail/utils/cuda/Utils.cuh>

#include <nanovdb/util/cuda/Util.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/types.h>

#include <hip/hip_cooperative_groups.h>

#include <algorithm>

namespace fvdb {

namespace detail {

namespace ops {

namespace {

namespace cg = cooperative_groups;

// ------------------------------------------
// Constant Memory for Gaussian Coefficients
// ------------------------------------------
__constant__ float cGauss[11] = {0.001028380123898387f,
                                 0.0075987582094967365f,
                                 0.036000773310661316f,
                                 0.10936068743467331f,
                                 0.21300552785396576f,
                                 0.26601171493530273f,
                                 0.21300552785396576f,
                                 0.10936068743467331f,
                                 0.036000773310661316f,
                                 0.0075987582094967365f,
                                 0.001028380123898387f};

// ------------------------------------------
// Block and Shared Memory Dimensions
// ------------------------------------------
#define BLOCK_X 16
#define BLOCK_Y 16
#define HALO    5

#define SHARED_X (BLOCK_X + 2 * HALO)
#define SHARED_Y (BLOCK_Y + 2 * HALO)

// For partial results after horizontal pass
#define CONV_X BLOCK_X
#define CONV_Y SHARED_Y

// ------------------------------------------
// Utility: Safe pixel fetch w/ zero padding
// ------------------------------------------
__device__ __forceinline__ float
getPixelValue(const float *img, int b, int c, int y, int x, int CH, int H, int W) {
    if (x < 0 || x >= W || y < 0 || y >= H) {
        return 0.0f;
    }
    return img[b * CH * H * W + c * H * W + y * W + x];
}

// ------------------------------------------
// Forward Kernel: Fused SSIM
//  - Two-pass convolution to get mu1, mu2,
//    sigma1_sq, sigma2_sq, sigma12, etc.
//  - Writes final SSIM map to ssim_map
//  - Optionally writes partial derivatives
//    to dm_dmu1, dm_dsigma1_sq, dm_dsigma12
// ------------------------------------------
__global__ void
fusedSSIMKernel(int localToGlobalOffset,
                int B,
                int H,
                int W,
                int CH,
                float C1,
                float C2,
                const float *__restrict__ img1,
                const float *__restrict__ img2,
                float *__restrict__ ssim_map,
                float *__restrict__ dm_dmu1,
                float *__restrict__ dm_dsigma1_sq,
                float *__restrict__ dm_dsigma12) {
    auto block = cg::this_thread_block();

    auto globalLinearGroupIndex = block.group_index().x + localToGlobalOffset;
    dim3 globalGroupDim((W + BLOCK_X - 1) / BLOCK_X, (H + BLOCK_Y - 1) / BLOCK_Y, B);
    dim3 globalGroupIndex(globalLinearGroupIndex % globalGroupDim.x,
                          (globalLinearGroupIndex / globalGroupDim.x) % globalGroupDim.y,
                          (globalLinearGroupIndex / (globalGroupDim.x * globalGroupDim.y)));

    const int bIdx    = globalGroupIndex.z; // batch index
    const int pix_y   = globalGroupIndex.y * BLOCK_Y + block.thread_index().y;
    const int pix_x   = globalGroupIndex.x * BLOCK_X + block.thread_index().x;
    const int pix_id  = pix_y * W + pix_x;
    const int num_pix = H * W;

    // Shared memory for the tile (img1, img2)
    __shared__ float sTile[SHARED_Y][SHARED_X][2];
    // After horizontal pass, store partial sums here
    // xconv[y][x] -> (sumX, sumX^2, sumY, sumY^2, sumXY)
    __shared__ float xconv[CONV_Y][CONV_X][5];

    // Each block processes B x C sub-batches. We loop over channels:
    for (int c = 0; c < CH; ++c) {
        // ------------------------------------------------------------
        // 1) Load (img1, img2) tile + halo into shared memory
        // ------------------------------------------------------------
        {
            const int tileSize = SHARED_Y * SHARED_X;
            const int threads  = BLOCK_X * BLOCK_Y;
            const int steps    = (tileSize + threads - 1) / threads;

            const int tileStartY = globalGroupIndex.y * BLOCK_Y;
            const int tileStartX = globalGroupIndex.x * BLOCK_X;

            for (int s = 0; s < steps; ++s) {
                int tid = s * threads + block.thread_rank();
                if (tid < tileSize) {
                    int local_y = tid / SHARED_X;
                    int local_x = tid % SHARED_X;
                    int gy      = tileStartY + local_y - HALO;
                    int gx      = tileStartX + local_x - HALO;

                    float X = getPixelValue(img1, bIdx, c, gy, gx, CH, H, W);
                    float Y = getPixelValue(img2, bIdx, c, gy, gx, CH, H, W);

                    sTile[local_y][local_x][0] = X;
                    sTile[local_y][local_x][1] = Y;
                }
            }
        }
        block.sync();

        // ------------------------------------------------------------
        // 2) Horizontal convolution (11x1) in shared memory
        //    We'll accumulate symmetrical pairs around center.
        // ------------------------------------------------------------
        {
            int ly = threadIdx.y;
            int lx = threadIdx.x + HALO; // skip left halo

            float sumX  = 0.f;
            float sumX2 = 0.f;
            float sumY  = 0.f;
            float sumY2 = 0.f;
            float sumXY = 0.f;

            // #pragma unroll for those 5 pairs
#pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                float w      = cGauss[HALO - d];
                float Xleft  = sTile[ly][lx - d][0];
                float Yleft  = sTile[ly][lx - d][1];
                float Xright = sTile[ly][lx + d][0];
                float Yright = sTile[ly][lx + d][1];

                sumX += (Xleft + Xright) * w;
                sumX2 += ((Xleft * Xleft) + (Xright * Xright)) * w;
                sumY += (Yleft + Yright) * w;
                sumY2 += ((Yleft * Yleft) + (Yright * Yright)) * w;
                sumXY += ((Xleft * Yleft) + (Xright * Yright)) * w;
            }
            // center
            {
                float centerX = sTile[ly][lx][0];
                float centerY = sTile[ly][lx][1];
                float wc      = cGauss[HALO];
                sumX += centerX * wc;
                sumX2 += (centerX * centerX) * wc;
                sumY += centerY * wc;
                sumY2 += (centerY * centerY) * wc;
                sumXY += (centerX * centerY) * wc;
            }

            // Write out partial sums
            xconv[ly][threadIdx.x][0] = sumX;
            xconv[ly][threadIdx.x][1] = sumX2;
            xconv[ly][threadIdx.x][2] = sumY;
            xconv[ly][threadIdx.x][3] = sumY2;
            xconv[ly][threadIdx.x][4] = sumXY;

            // Possibly handle second row in same warp
            int ly2 = ly + BLOCK_Y;
            if (ly2 < CONV_Y) {
                sumX  = 0.f;
                sumX2 = 0.f;
                sumY  = 0.f;
                sumY2 = 0.f;
                sumXY = 0.f;

#pragma unroll
                for (int d = 1; d <= HALO; ++d) {
                    float w      = cGauss[HALO - d];
                    float Xleft  = sTile[ly2][lx - d][0];
                    float Yleft  = sTile[ly2][lx - d][1];
                    float Xright = sTile[ly2][lx + d][0];
                    float Yright = sTile[ly2][lx + d][1];

                    sumX += (Xleft + Xright) * w;
                    sumX2 += ((Xleft * Xleft) + (Xright * Xright)) * w;
                    sumY += (Yleft + Yright) * w;
                    sumY2 += ((Yleft * Yleft) + (Yright * Yright)) * w;
                    sumXY += ((Xleft * Yleft) + (Xright * Yright)) * w;
                }
                // center
                {
                    float cx = sTile[ly2][lx][0];
                    float cy = sTile[ly2][lx][1];
                    float wc = cGauss[HALO];
                    sumX += cx * wc;
                    sumX2 += (cx * cx) * wc;
                    sumY += cy * wc;
                    sumY2 += (cy * cy) * wc;
                    sumXY += (cx * cy) * wc;
                }
                xconv[ly2][threadIdx.x][0] = sumX;
                xconv[ly2][threadIdx.x][1] = sumX2;
                xconv[ly2][threadIdx.x][2] = sumY;
                xconv[ly2][threadIdx.x][3] = sumY2;
                xconv[ly2][threadIdx.x][4] = sumXY;
            }
        }
        block.sync();

        // ------------------------------------------------------------
        // 3) Vertical convolution (1x11) + final SSIM
        // ------------------------------------------------------------
        {
            int ly = threadIdx.y + HALO;
            int lx = threadIdx.x;

            float out0 = 0.f, out1 = 0.f, out2 = 0.f, out3 = 0.f, out4 = 0.f;

#pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                float w    = cGauss[HALO - d];
                float *top = xconv[ly - d][lx];
                float *bot = xconv[ly + d][lx];

                out0 += (top[0] + bot[0]) * w;
                out1 += (top[1] + bot[1]) * w;
                out2 += (top[2] + bot[2]) * w;
                out3 += (top[3] + bot[3]) * w;
                out4 += (top[4] + bot[4]) * w;
            }
            // center
            {
                float wC   = cGauss[HALO];
                float *ctr = xconv[ly][lx];
                out0 += ctr[0] * wC;
                out1 += ctr[1] * wC;
                out2 += ctr[2] * wC;
                out3 += ctr[3] * wC;
                out4 += ctr[4] * wC;
            }

            if (pix_x < W && pix_y < H) {
                float mu1    = out0;
                float mu2    = out2;
                float mu1_sq = mu1 * mu1;
                float mu2_sq = mu2 * mu2;

                float sigma1_sq = out1 - mu1_sq;
                float sigma2_sq = out3 - mu2_sq;
                float sigma12   = out4 - mu1 * mu2;

                float A  = mu1_sq + mu2_sq + C1;
                float B  = sigma1_sq + sigma2_sq + C2;
                float C_ = 2.f * mu1 * mu2 + C1;
                float D_ = 2.f * sigma12 + C2;

                float val = (C_ * D_) / (A * B);

                int global_idx       = bIdx * CH * num_pix + c * num_pix + pix_id;
                ssim_map[global_idx] = val;

                if (dm_dmu1) {
                    // partial derivatives
                    float d_m_dmu1 =
                        ((mu2 * 2.f * D_) / (A * B) - (mu2 * 2.f * C_) / (A * B) -
                         (mu1 * 2.f * C_ * D_) / (A * A * B) + (mu1 * 2.f * C_ * D_) / (A * B * B));
                    float d_m_dsigma1_sq = (-C_ * D_) / (A * B * B);
                    float d_m_dsigma12   = (2.f * C_) / (A * B);

                    dm_dmu1[global_idx]       = d_m_dmu1;
                    dm_dsigma1_sq[global_idx] = d_m_dsigma1_sq;
                    dm_dsigma12[global_idx]   = d_m_dsigma12;
                }
            }
        }
    }
}

// ------------------------------------------
// Backward Kernel: Apply chain rule to get
//    dL/d(img1) from partial derivatives
//    (dm_dmu1, dm_dsigma1_sq, dm_dsigma12)
//    and dL/dmap (the gradient from above).
// ------------------------------------------
__global__ void
fusedSSIMBackwardKernel(int localToGlobalOffset,
                        int B,
                        int H,
                        int W,
                        int CH,
                        float C1,
                        float C2,
                        const float *__restrict__ img1,
                        const float *__restrict__ img2,
                        const float *__restrict__ dL_dmap,
                        float *__restrict__ dL_dimg1,
                        const float *__restrict__ dm_dmu1,
                        const float *__restrict__ dm_dsigma1_sq,
                        const float *__restrict__ dm_dsigma12) {
    auto block = cg::this_thread_block();

    auto globalLinearGroupIndex = block.group_index().x + localToGlobalOffset;
    dim3 globalGroupDim((W + BLOCK_X - 1) / BLOCK_X, (H + BLOCK_Y - 1) / BLOCK_Y, B);
    dim3 globalGroupIndex(globalLinearGroupIndex % globalGroupDim.x,
                          (globalLinearGroupIndex / globalGroupDim.x) % globalGroupDim.y,
                          (globalLinearGroupIndex / (globalGroupDim.x * globalGroupDim.y)));

    const int bIdx    = globalGroupIndex.z; // batch index
    const int pix_y   = globalGroupIndex.y * BLOCK_Y + block.thread_index().y;
    const int pix_x   = globalGroupIndex.x * BLOCK_X + block.thread_index().x;
    const int pix_id  = pix_y * W + pix_x;
    const int num_pix = H * W;

    // Shared memory for the fused data:
    // [0]: dm_dmu1*dL, [1]: dm_dsigma1_sq*dL, [2]: dm_dsigma12*dL
    __shared__ float sData[3][SHARED_Y][SHARED_X];
    __shared__ float sScratch[CONV_Y][CONV_X][3];

    for (int c = 0; c < CH; ++c) {
        float p1 = 0.f, p2 = 0.f;
        if (pix_x < W && pix_y < H) {
            p1 = getPixelValue(img1, bIdx, c, pix_y, pix_x, CH, H, W);
            p2 = getPixelValue(img2, bIdx, c, pix_y, pix_x, CH, H, W);
        }

        // (1) Load + fuse multiplication
        {
            const int start_y = globalGroupIndex.y * BLOCK_Y;
            const int start_x = globalGroupIndex.x * BLOCK_X;

            int tid          = threadIdx.y * blockDim.x + threadIdx.x;
            int warp_id      = tid / 32;
            int lane_id      = tid % 32;
            int totalThreads = BLOCK_X * BLOCK_Y;
            int num_warps    = (totalThreads + 31) / 32;

            for (int row = warp_id; row < SHARED_Y; row += num_warps) {
                int gy = start_y + row - HALO;
                for (int col = lane_id; col < SHARED_X; col += 32) {
                    int gx = start_x + col - HALO;

                    float chain = getPixelValue(dL_dmap, bIdx, c, gy, gx, CH, H, W);
                    float vmu   = getPixelValue(dm_dmu1, bIdx, c, gy, gx, CH, H, W);
                    float vs1   = getPixelValue(dm_dsigma1_sq, bIdx, c, gy, gx, CH, H, W);
                    float vs12  = getPixelValue(dm_dsigma12, bIdx, c, gy, gx, CH, H, W);

                    sData[0][row][col] = vmu * chain;
                    sData[1][row][col] = vs1 * chain;
                    sData[2][row][col] = vs12 * chain;
                }
            }
        }
        block.sync();

        // (2) Horizontal pass
        {
            int ly = threadIdx.y;
            int lx = threadIdx.x + HALO;

            for (int pass = 0; pass < 2; ++pass) {
                int yy = ly + pass * BLOCK_Y;
                if (yy < CONV_Y) {
                    float accum0 = 0.f, accum1 = 0.f, accum2 = 0.f;

#pragma unroll
                    for (int d = 1; d <= HALO; ++d) {
                        float w     = cGauss[HALO - d];
                        float left0 = sData[0][yy][lx - d];
                        float left1 = sData[1][yy][lx - d];
                        float left2 = sData[2][yy][lx - d];

                        float right0 = sData[0][yy][lx + d];
                        float right1 = sData[1][yy][lx + d];
                        float right2 = sData[2][yy][lx + d];

                        accum0 += (left0 + right0) * w;
                        accum1 += (left1 + right1) * w;
                        accum2 += (left2 + right2) * w;
                    }
                    // center
                    {
                        float wc = cGauss[HALO];
                        float c0 = sData[0][yy][lx];
                        float c1 = sData[1][yy][lx];
                        float c2 = sData[2][yy][lx];
                        accum0 += c0 * wc;
                        accum1 += c1 * wc;
                        accum2 += c2 * wc;
                    }

                    sScratch[yy][threadIdx.x][0] = accum0;
                    sScratch[yy][threadIdx.x][1] = accum1;
                    sScratch[yy][threadIdx.x][2] = accum2;
                }
            }
        }
        block.sync();

        // (3) Vertical pass -> finalize dL/d(img1)
        if (pix_x < W && pix_y < H) {
            int ly = threadIdx.y + HALO;
            int lx = threadIdx.x;

            float sum0 = 0.f, sum1 = 0.f, sum2 = 0.f;

#pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                float w    = cGauss[HALO - d];
                float *top = sScratch[ly - d][lx];
                float *bot = sScratch[ly + d][lx];

                sum0 += (top[0] + bot[0]) * w;
                sum1 += (top[1] + bot[1]) * w;
                sum2 += (top[2] + bot[2]) * w;
            }
            // center
            {
                float wc   = cGauss[HALO];
                float *ctr = sScratch[ly][lx];
                sum0 += ctr[0] * wc;
                sum1 += ctr[1] * wc;
                sum2 += ctr[2] * wc;
            }

            // final accumulation
            float dL_dpix = sum0 + (2.f * p1) * sum1 + (p2)*sum2;

            int out_idx       = bIdx * CH * num_pix + c * num_pix + pix_id;
            dL_dimg1[out_idx] = dL_dpix;
        }
        block.sync();
    }
}

} // namespace

// ------------------------------------------
// PyTorch Interface (Forward)
//   Returns (ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12).
//   If train=false, derivative Tensors are empty.
// ------------------------------------------
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fusedSSIMCUDA(
    double C1, double C2, const torch::Tensor &img1, const torch::Tensor &img2, bool train) {
    const at::cuda::OptionalCUDAGuard device_guard(device_of(img1));
    const auto stream = at::cuda::getCurrentCUDAStream(img1.device().index());
    int B             = img1.size(0);
    int CH            = img1.size(1);
    int H             = img1.size(2);
    int W             = img1.size(3);

    TORCH_CHECK_VALUE(img1.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");
    TORCH_CHECK_VALUE(img2.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");

    // Launch config
    dim3 grid(((W + BLOCK_X - 1) / BLOCK_X) * ((H + BLOCK_Y - 1) / BLOCK_Y) * B);
    dim3 block(BLOCK_X, BLOCK_Y);

    // Output SSIM map
    auto ssim_map = torch::empty_like(img1, img1.options()).contiguous();

    // Optionally allocate derivative Tensors
    auto dm_dmu1       = train ? torch::empty_like(img1) : torch::empty({0}, img1.options());
    auto dm_dsigma1_sq = train ? torch::empty_like(img1) : torch::empty({0}, img1.options());
    auto dm_dsigma12   = train ? torch::empty_like(img1) : torch::empty({0}, img1.options());

    fusedSSIMKernel<<<grid, block, 0, stream>>>(0,
                                                B,
                                                H,
                                                W,
                                                CH,
                                                static_cast<float>(C1),
                                                static_cast<float>(C2),
                                                img1.contiguous().const_data_ptr<float>(),
                                                img2.contiguous().const_data_ptr<float>(),
                                                ssim_map.data_ptr<float>(),
                                                train ? dm_dmu1.data_ptr<float>() : nullptr,
                                                train ? dm_dsigma1_sq.data_ptr<float>() : nullptr,
                                                train ? dm_dsigma12.data_ptr<float>() : nullptr);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return std::make_tuple(ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12);
}

// ------------------------------------------
// PyTorch Interface (Backward)
//   Takes the gradient wrt the SSIM map and
//   the partial derivatives from forward;
//   returns dL/d(img1).
// ------------------------------------------
torch::Tensor
fusedSSIMBackwardCUDA(double C1,
                      double C2,
                      const torch::Tensor &img1,
                      const torch::Tensor &img2,
                      const torch::Tensor &dL_dmap,
                      const torch::Tensor &dm_dmu1,
                      const torch::Tensor &dm_dsigma1_sq,
                      const torch::Tensor &dm_dsigma12) {
    const at::cuda::OptionalCUDAGuard device_guard(device_of(img1));
    const auto stream = at::cuda::getCurrentCUDAStream(img1.device().index());
    int B             = img1.size(0);
    int CH            = img1.size(1);
    int H             = img1.size(2);
    int W             = img1.size(3);

    TORCH_CHECK_VALUE(img1.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");
    TORCH_CHECK_VALUE(img2.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");

    auto dL_dimg1 = torch::empty_like(img1);

    dim3 grid(((W + BLOCK_X - 1) / BLOCK_X) * ((H + BLOCK_Y - 1) / BLOCK_Y) * B);
    dim3 block(BLOCK_X, BLOCK_Y);

    fusedSSIMBackwardKernel<<<grid, block, 0, stream>>>(
        0,
        B,
        H,
        W,
        CH,
        static_cast<float>(C1),
        static_cast<float>(C2),
        img1.contiguous().const_data_ptr<float>(),
        img2.contiguous().const_data_ptr<float>(),
        dL_dmap.contiguous().const_data_ptr<float>(),
        dL_dimg1.data_ptr<float>(),
        dm_dmu1.contiguous().const_data_ptr<float>(),
        dm_dsigma1_sq.contiguous().const_data_ptr<float>(),
        dm_dsigma12.contiguous().const_data_ptr<float>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return dL_dimg1;
}

// ------------------------------------------
// PyTorch Interface (Forward)
//   Returns (ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12).
//   If train=false, derivative Tensors are empty.
// ------------------------------------------
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fusedSSIMPrivateUse1(
    double C1, double C2, const torch::Tensor &img1, const torch::Tensor &img2, bool train) {
    int B  = img1.size(0);
    int CH = img1.size(1);
    int H  = img1.size(2);
    int W  = img1.size(3);

    TORCH_CHECK_VALUE(img1.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");
    TORCH_CHECK_VALUE(img2.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");

    // Output SSIM map
    auto ssim_map = torch::empty_like(img1, img1.options()).contiguous();

    // Optionally allocate derivative Tensors
    auto dm_dmu1       = train ? torch::empty_like(img1) : torch::empty({0}, img1.options());
    auto dm_dsigma1_sq = train ? torch::empty_like(img1) : torch::empty({0}, img1.options());
    auto dm_dsigma12   = train ? torch::empty_like(img1) : torch::empty({0}, img1.options());

    auto img1_ = img1.contiguous();
    auto img2_ = img2.contiguous();

    const auto globalBlockCount = ((W + BLOCK_X - 1) / BLOCK_X) * ((H + BLOCK_Y - 1) / BLOCK_Y) * B;
    for (const auto deviceId: c10::irange(c10::cuda::device_count())) {
        C10_CUDA_CHECK(hipSetDevice(deviceId));
        auto stream = c10::cuda::getCurrentCUDAStream(deviceId);

        constexpr size_t kAlignment = kPageSize / (sizeof(float) * BLOCK_X * BLOCK_Y);
        int localBlockOffset, localBlockCount;
        std::tie(localBlockOffset, localBlockCount) =
            deviceAlignedChunk(kAlignment, globalBlockCount, deviceId);

        if (localBlockCount) {
            auto localElementOffset = localBlockOffset * BLOCK_X * BLOCK_Y * CH;
            auto localElementCount  = localBlockCount * BLOCK_X * BLOCK_Y * CH;
            if (localElementOffset + localElementCount > img1_.numel()) {
                localElementOffset = std::min(localElementOffset, static_cast<int>(img1_.numel()));
                localElementCount  = std::min(localElementCount,
                                             static_cast<int>(img1_.numel()) - localElementOffset);
            }
            nanovdb::util::cuda::memPrefetchAsync(img1_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);
            nanovdb::util::cuda::memPrefetchAsync(img2_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);

            // Launch config
            dim3 grid(localBlockCount);
            dim3 block(BLOCK_X, BLOCK_Y);

            fusedSSIMKernel<<<grid, block, 0, stream>>>(
                localBlockOffset,
                B,
                H,
                W,
                CH,
                static_cast<float>(C1),
                static_cast<float>(C2),
                img1_.const_data_ptr<float>(),
                img2_.const_data_ptr<float>(),
                ssim_map.data_ptr<float>(),
                train ? dm_dmu1.data_ptr<float>() : nullptr,
                train ? dm_dsigma1_sq.data_ptr<float>() : nullptr,
                train ? dm_dsigma12.data_ptr<float>() : nullptr);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
    }

    mergeStreams();

    return std::make_tuple(ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12);
}

// ------------------------------------------
// PyTorch Interface (Backward)
//   Takes the gradient wrt the SSIM map and
//   the partial derivatives from forward;
//   returns dL/d(img1).
// ------------------------------------------
torch::Tensor
fusedSSIMBackwardPrivateUse1(double C1,
                             double C2,
                             const torch::Tensor &img1,
                             const torch::Tensor &img2,
                             const torch::Tensor &dL_dmap,
                             const torch::Tensor &dm_dmu1,
                             const torch::Tensor &dm_dsigma1_sq,
                             const torch::Tensor &dm_dsigma12) {
    int B  = img1.size(0);
    int CH = img1.size(1);
    int H  = img1.size(2);
    int W  = img1.size(3);

    TORCH_CHECK_VALUE(img1.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");
    TORCH_CHECK_VALUE(img2.scalar_type() == torch::kFloat,
                      "Fused SSIM only supports float32 images");

    auto dL_dimg1 = torch::empty_like(img1);

    auto img1_ = img1.contiguous();
    auto img2_ = img2.contiguous();

    auto dL_dmap_       = dL_dmap.contiguous();
    auto dm_dmu1_       = dm_dmu1.contiguous();
    auto dm_dsigma1_sq_ = dm_dsigma1_sq.contiguous();
    auto dm_dsigma12_   = dm_dsigma12.contiguous();

    const auto globalBlockCount = ((W + BLOCK_X - 1) / BLOCK_X) * ((H + BLOCK_Y - 1) / BLOCK_Y) * B;
    for (const auto deviceId: c10::irange(c10::cuda::device_count())) {
        C10_CUDA_CHECK(hipSetDevice(deviceId));
        auto stream = c10::cuda::getCurrentCUDAStream(deviceId);

        constexpr size_t kAlignment = kPageSize / (sizeof(float) * BLOCK_X * BLOCK_Y);
        int localBlockOffset, localBlockCount;
        std::tie(localBlockOffset, localBlockCount) =
            deviceAlignedChunk(kAlignment, globalBlockCount, deviceId);

        if (localBlockCount) {
            auto localElementOffset = localBlockOffset * BLOCK_X * BLOCK_Y * CH;
            auto localElementCount  = localBlockCount * BLOCK_X * BLOCK_Y * CH;
            if (localElementOffset + localElementCount > img1_.numel()) {
                localElementOffset = std::min(localElementOffset, static_cast<int>(img1_.numel()));
                localElementCount  = std::min(localElementCount,
                                             static_cast<int>(img1_.numel()) - localElementOffset);
            }
            nanovdb::util::cuda::memPrefetchAsync(img1_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);
            nanovdb::util::cuda::memPrefetchAsync(img2_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);
            nanovdb::util::cuda::memPrefetchAsync(dL_dmap_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);
            nanovdb::util::cuda::memPrefetchAsync(dm_dmu1_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);
            nanovdb::util::cuda::memPrefetchAsync(dm_dsigma1_sq_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);
            nanovdb::util::cuda::memPrefetchAsync(dm_dsigma12_.const_data_ptr<float>() +
                                                      localElementOffset,
                                                  localElementCount * sizeof(float),
                                                  deviceId,
                                                  stream);

            // Launch config
            dim3 grid(localBlockCount);
            dim3 block(BLOCK_X, BLOCK_Y);

            fusedSSIMBackwardKernel<<<grid, block, 0, stream>>>(
                localBlockOffset,
                B,
                H,
                W,
                CH,
                static_cast<float>(C1),
                static_cast<float>(C2),
                img1_.const_data_ptr<float>(),
                img2_.const_data_ptr<float>(),
                dL_dmap_.const_data_ptr<float>(),
                dL_dimg1.data_ptr<float>(),
                dm_dmu1_.const_data_ptr<float>(),
                dm_dsigma1_sq_.const_data_ptr<float>(),
                dm_dsigma12_.const_data_ptr<float>());
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
    }

    mergeStreams();

    return dL_dimg1;
}

} // namespace ops

} // namespace detail

} // namespace fvdb
